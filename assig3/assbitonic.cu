#include "hip/hip_runtime.h"
/**
 *
 */

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/types.h>
#include <time.h>

#include "common.h"
#include <hip/hip_runtime.h>

/* allusion to internal functions */

static void sort_cpu_kernel(unsigned int *matrix, unsigned int matrix_size);
__global__ static void sort_cuda_kernel(unsigned int *__restrict__ matrix, unsigned int matrix_size, unsigned int k, unsigned int j);
static double get_delta_time(void);

/**
 *   main program
 */

int main(int argc, char **argv) {

  printf("%s Starting...\n", argv[0]);
  if (sizeof(unsigned int) != (size_t)4)
    return 1; // it fails with prejudice if an integer does not have 4 bytes

  /* get argv of input file name */
  if (argc != 2) {
    printf("Invalid program usage!\nRun with:\n\t./bitonic <input file name>\n");
    return 2;
  }

  /* check if file exists */
  FILE *fd = fopen(argv[1], "r");
  if (fd == NULL) {
    printf("ERROR: Could not open the file (%s)\n", argv[1]);
    return 3;
  }

  /* read the first line (first line has the sample size) */
  u_int32_t n;
  if (fread(&n, sizeof(unsigned int), 1, fd) != 1) {
    printf("ERROR: reading the first row\n");
    return 4;
  }
  if ((n & (n - 1)) != 0) {
    printf("ERROR: number of numbers is not a power of two!\n");
    return 5;
  }

  /* allocate host matrix with and fill it with file numbers */
  get_delta_time();
  unsigned int *host_unsorted = (unsigned int *)malloc(n * sizeof(unsigned int));
  if (fread(host_unsorted, sizeof(unsigned int), n, fd) != n) {
    printf("ERROR: in reading the numbers from the file\n");
    return 6;
  }
  printf("Reading %d values from the file took %.3e seconds\n", n, get_delta_time());

  /* set up the device */

  int dev = 0;

  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  /* create the remaining memory areas in host and device memory where the disk sectors data and sector numbers will be stored */

  size_t matrix_size = n * sizeof(unsigned int);
  unsigned int *host_sorted, *host_device_sorted;
  unsigned int *device_matrix;

  if (matrix_size > (size_t)5e9) {
    fprintf(stderr, "The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
    exit(1);
  }

  host_sorted = (unsigned int *)malloc(matrix_size);
  host_device_sorted = (unsigned int *)malloc(matrix_size);
  CHECK(hipMalloc((void **)&device_matrix, matrix_size));

  unsigned int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;
  // for (int i = 0; i < 0; i++) {
  //  double avg = 0;
  // for (int j = 0; j < 10; j++) {

  /* copy the host data to the device memory */

  (void)get_delta_time();
  CHECK(hipMemcpy(device_matrix, host_unsorted, matrix_size, hipMemcpyHostToDevice));
  printf("The transfer of %ld bytes from the host to the device took %.3e seconds\n", matrix_size, get_delta_time());

  /* run the computational kernel (gpu) */

  blockDimX = 32; // optimize!
  blockDimY = 32; // optimize!
  blockDimZ = 1;  // do not change!
  gridDimX = 16;  // optimize!
  gridDimY = 32;  // optimize!
  gridDimZ = 1;   // do not change!

  /*
 blockDimX = 4; // optimize!
 blockDimY = 4;     // optimize!
 blockDimZ = 1;     // do not change!
 gridDimX = 1;      // optimize!
 gridDimY = 1;      // optimize!
 gridDimZ = 1;      // do not change!
 */

  dim3 grid(gridDimX, gridDimY, gridDimZ);
  dim3 block(blockDimX, blockDimY, blockDimZ);

  if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != n / 2) {
    printf("Wrong configuration! %d != %d\n", (gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ), n / 2);
    return 1;
  }
  uint k, kk, j;
  (void)get_delta_time();
  for (k = 2, kk = 1; k <= n; k *= 2, kk++) { // for each iteration
    for (j = k / 2; j >= 1; j /= 2) {         // for each step
      sort_cuda_kernel<<<grid, block>>>(device_matrix, matrix_size, k, j);
      CHECK(hipDeviceSynchronize()); // wait for kernel to finish
    }
  }
  CHECK(hipGetLastError()); // check for kernel errors
  /*avg += get_delta_time();
}
*/
  printf("\nThe CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time());

  /* copy kernel result back to host side */
  (void)get_delta_time();
  CHECK(hipMemcpy(host_device_sorted, device_matrix, matrix_size, hipMemcpyDeviceToHost));
  printf("The transfer of %ld bytes from the device to the host took %.3e seconds\n", (long)matrix_size, get_delta_time());

  /* free device global memory */

  CHECK(hipFree(device_matrix));

  /* reset the device */

  CHECK(hipDeviceReset());

  /* compute the modified sector data on the CPU */

  // memcpy(host_sorted, host_unsorted, n);
  (void)get_delta_time();
  sort_cpu_kernel(host_unsorted, n);
  printf("\nThe cpu kernel took %.3e seconds to run (single core)\n", get_delta_time());

  /* verify results */

  // printf("SORTED?\n");
  for (int i = 0; i < n - 1; i++) {
    // printf("%2d: %d\n", i, host_device_sorted[i]);
    if (host_device_sorted[i + 1] < host_device_sorted[i]) {
      // printf("!");
      printf("ERROR: CPU NOT SORTED\n");
      return 8;
    }
  }

  printf("All is well!\n");

  /* free host memory */

  free(host_unsorted);
  free(host_sorted);
  free(host_device_sorted);

  return 0;
}

static void sort_cpu_kernel(unsigned int *arr, unsigned int n) {
  uint i, j, k, l;
  uint kk;
  uint temp;

  for (k = 2, kk = 1; k <= n; k *= 2, kk++) { // for each iteration
    // printf("ITER %d, %d subsequences\n", kk, k);
    for (j = k / 2; j >= 1; j /= 2) { // for each step
      // printf("\tSTEP %d\n", j);
      for (i = 0; i < n; i += j << 1) { // for each block
        uint sort = ((i / k)) & 0x1;    // 0 for ascending block and 1 for descending
        // printf("\t\tblock start: %d %s\n", i, sort ? "<" : ">");
        for (l = i; l < i + j; l++) {
          // printf("\t\t\t(%d, %d)\n", l, l + j);
          if ((!sort && arr[l] > arr[l + j]) || (sort && arr[l] < arr[l + j])) {
            temp = arr[l];
            arr[l] = arr[l + j];
            arr[l + j] = temp;
          }
        }
      }
    }
  }
}

/**
 * arr -> sequence to sort
 * n -> size of the sequence
 * k -> current iteration
 * j -> current step
 **/
__global__ static void sort_cuda_kernel(unsigned int *__restrict__ arr, unsigned int n, unsigned int k, unsigned int j) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = row * 512 + col;

  uint i, l, temp;

  // printf("\t%2d: ITER %d STEP %d\n", idx, k, j);

  // for (i = 0; i < n; i += j << 1) { // for each block
  i = idx / (j) * (j << 1);    // currespondent block
  uint sort = ((i / k)) & 0x1; // 0 for ascending block and 1 for descending
                               // printf("\t\tblock start: %d %s\n", i, sort ? "<" : ">");
  l = i + idx % j;
  //  for (l = i; l < i + j; l++) {
  // printf("\t\t%2d: %d(%d, %d)\n", idx, j, l, l + j);
  if ((!sort && arr[l] > arr[l + j]) || (sort && arr[l] < arr[l + j])) {
    temp = arr[l];
    arr[l] = arr[l + j];
    arr[l + j] = temp;
  }
  //}
  //}
  // printf("(%d %d %d)\n", row, col, idx);
}

/*
 * CODE USED TO TEST THE CPU MATRIX MULTIPLICATION PRIMITIVE
 *
  unsigned int *a, *b, *c;
  a = (unsigned int *)malloc(9 * sizeof(unsigned int));
  b = (unsigned int *)malloc(9 * sizeof(unsigned int));
  c = (unsigned int *)malloc(9 * sizeof(unsigned int));
  for (int i = 0; i < 9; i++)
    a[i] = i + 1;
  for (int i = 0; i < 9; i++)
    b[i] = 9 - i;

  printf("A:\n");
  for (int i = 0; i < 9; i++) {
    printf("%d ", a[i]);
    if ((i + 1) % 3 == 0)
      printf("\n");
  }
  printf("\nB:\n");
  for (int i = 0; i < 9; i++) {
    printf("%d ", b[i]);
    if ((i + 1) % 3 == 0)
      printf("\n");
  }
  matrix_mult_cpu_kernel(a, b, c, 9 * sizeof(unsigned int));
  printf("\nC:\n");
  for (int i = 0; i < 9; i++) {
    printf("%d ", c[i]);
    if ((i + 1) % 3 == 0)
      printf("\n");
  }

  exit(0);
  */
static double get_delta_time(void) {
  static struct timespec t0, t1;

  t0 = t1;
  if (clock_gettime(CLOCK_MONOTONIC, &t1) != 0) {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
