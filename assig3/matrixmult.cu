#include "hip/hip_runtime.h"
/**
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

#ifndef SQUARE_MATRIX_SIZE
#define SQUARE_MATRIX_SIZE 1024
#endif

/* allusion to internal functions */

static void matrix_mult_cpu_kernel(unsigned int *matrix_A, unsigned int *matrix_B,
                                   unsigned int *matrix_C, unsigned int matrix_size);
__global__ static void matrix_mult_cuda_kernel(unsigned int *__restrict__ matrix_A, unsigned int *__restrict__ matrix_B,
                                               unsigned int *__restrict__ matrix_C, unsigned int matrix_size);
static double get_delta_time(void);

/**
 *   main program
 */

int main(int argc, char **argv) {

  printf("%s Starting...\n", argv[0]);
  if (sizeof(unsigned int) != (size_t)4)
    return 1; // it fails with prejudice if an integer does not have 4 bytes

  /* set up the device */

  int dev = 0;

  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  /* create memory areas in host and device memory where the disk sectors data and sector numbers will be stored */

  size_t matrix_size = SQUARE_MATRIX_SIZE * SQUARE_MATRIX_SIZE * sizeof(unsigned int);
  unsigned int *host_matrix_A, *host_matrix_B, *host_matrix_C, *host_device_matrix_C;
  unsigned int *device_matrix_A, *device_matrix_B, *device_matrix_C;

  if (matrix_size * 2 > (size_t)5e9) {
    fprintf(stderr, "The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
    exit(1);
  }

  host_matrix_A = (unsigned int *)malloc(matrix_size);
  host_matrix_B = (unsigned int *)malloc(matrix_size);
  host_matrix_C = (unsigned int *)malloc(matrix_size);
  host_device_matrix_C = (unsigned int *)malloc(matrix_size);
  CHECK(hipMalloc((void **)&device_matrix_A, matrix_size));
  CHECK(hipMalloc((void **)&device_matrix_B, matrix_size));
  CHECK(hipMalloc((void **)&device_matrix_C, matrix_size));

  /* initialize the host data */

  int i;

  printf("Start attribution of random data\n");
  (void)get_delta_time();
  srand(0xCCE2021);
  for (i = 0; i < SQUARE_MATRIX_SIZE * SQUARE_MATRIX_SIZE; i++) {
    host_matrix_A[i] = rand() & 0xFF;
    host_matrix_B[i] = rand() & 0xFF;
  }
  printf("The initialization of host data took %.3e seconds\n", get_delta_time());

  unsigned int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;
  for (int i = 0; i < 6; i++) {
    double avg = 0;
    for (int j = 0; j < 10; j++) {

      /* copy the host data to the device memory */

      (void)get_delta_time();
      CHECK(hipMemcpy(device_matrix_A, host_matrix_A, matrix_size, hipMemcpyHostToDevice));
      CHECK(hipMemcpy(device_matrix_B, host_matrix_B, matrix_size, hipMemcpyHostToDevice));
      /*printf("The transfer of %ld bytes from the host to the device took %.3e seconds\n",
             (long)sector_data_size + (long)sector_number_size, get_delta_time());*/

      /* run the computational kernel
         as an example, N_SECTORS threads are launched where each thread deals with one sector */

      blockDimX = 1 << i;       // optimize!
      blockDimY = 1 << i;       // optimize!
      blockDimZ = 1 << 0;       // do not change!
      gridDimX = 1 << (10 - i); // optimize!
      gridDimY = 1 << (10 - i); // optimize!
      gridDimZ = 1 << 0;        // do not change!

      dim3 grid(gridDimX, gridDimY, gridDimZ);
      dim3 block(blockDimX, blockDimY, blockDimZ);

      if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != SQUARE_MATRIX_SIZE * SQUARE_MATRIX_SIZE) {
        printf("Wrong configuration! %d != %d\n",
               (gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ), SQUARE_MATRIX_SIZE * SQUARE_MATRIX_SIZE);
        return 1;
      }
      (void)get_delta_time();
      matrix_mult_cuda_kernel<<<grid, block>>>(device_matrix_A, device_matrix_B, device_matrix_C, matrix_size);
      CHECK(hipDeviceSynchronize()); // wait for kernel to finish
      CHECK(hipGetLastError());      // check for kernel errors
      avg += get_delta_time();
    }
    printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
           gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, avg / 10);
  }

  /* copy kernel result back to host side */
  (void)get_delta_time();
  CHECK(hipMemcpy(host_device_matrix_C, device_matrix_C, matrix_size, hipMemcpyDeviceToHost));
  printf("The transfer of %ld bytes from the device to the host took %.3e seconds\n", (long)matrix_size, get_delta_time());

  /* free device global memory */

  CHECK(hipFree(device_matrix_A));
  CHECK(hipFree(device_matrix_B));
  CHECK(hipFree(device_matrix_C));

  /* reset the device */

  CHECK(hipDeviceReset());

  /* compute the modified sector data on the CPU */

  (void)get_delta_time();
  matrix_mult_cpu_kernel(host_matrix_A, host_matrix_B, host_matrix_C, matrix_size);
  printf("The cpu kernel took %.3e seconds to run (single core)\n", get_delta_time());

  /* compare results */

  for (i = 0; i < SQUARE_MATRIX_SIZE * SQUARE_MATRIX_SIZE; i++)
    if (host_device_matrix_C[i] != host_matrix_C[i]) {
      printf("Mismatch in elem %d\n", i);
      exit(1);
    }
  printf("All is well!\n");

  /* free host memory */

  free(host_matrix_A);
  free(host_matrix_B);
  free(host_matrix_C);

  return 0;
}

static void matrix_mult_cpu_kernel(unsigned int *matrix_A, unsigned int *matrix_B,
                                   unsigned int *matrix_C, unsigned int sector_size) {
  long sum = 0, a, b;
  for (int cy = 0; cy < SQUARE_MATRIX_SIZE; cy++) {
    for (int cx = 0; cx < SQUARE_MATRIX_SIZE; cx++) {
      sum = 0;
      for (int i = 0; i < SQUARE_MATRIX_SIZE; i++) {
        a = matrix_A[cy * SQUARE_MATRIX_SIZE + i];
        b = matrix_B[i * SQUARE_MATRIX_SIZE + cx];
        sum += a * b;
      }
      matrix_C[cy * SQUARE_MATRIX_SIZE + cx] = sum;
    }
  }
}

__global__ static void matrix_mult_cuda_kernel(unsigned int *__restrict__ matrix_A, unsigned int *__restrict__ matrix_B,
                                               unsigned int *__restrict__ matrix_C, unsigned int matrix_size) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int sum = 0;
  for (int k = 0; k < SQUARE_MATRIX_SIZE; k++) {
    sum += matrix_A[row * SQUARE_MATRIX_SIZE + k] * matrix_B[k * SQUARE_MATRIX_SIZE + col];
  }
  matrix_C[row * SQUARE_MATRIX_SIZE + col] = sum;
}

static double get_delta_time(void) {
  static struct timespec t0, t1;

  t0 = t1;
  if (clock_gettime(CLOCK_MONOTONIC, &t1) != 0) {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}

/*
 * CODE USED TO TEST THE CPU MATRIX MULTIPLICATION PRIMITIVE
 *
  unsigned int *a, *b, *c;
  a = (unsigned int *)malloc(9 * sizeof(unsigned int));
  b = (unsigned int *)malloc(9 * sizeof(unsigned int));
  c = (unsigned int *)malloc(9 * sizeof(unsigned int));
  for (int i = 0; i < 9; i++)
    a[i] = i + 1;
  for (int i = 0; i < 9; i++)
    b[i] = 9 - i;

  printf("A:\n");
  for (int i = 0; i < 9; i++) {
    printf("%d ", a[i]);
    if ((i + 1) % 3 == 0)
      printf("\n");
  }
  printf("\nB:\n");
  for (int i = 0; i < 9; i++) {
    printf("%d ", b[i]);
    if ((i + 1) % 3 == 0)
      printf("\n");
  }
  matrix_mult_cpu_kernel(a, b, c, 9 * sizeof(unsigned int));
  printf("\nC:\n");
  for (int i = 0; i < 9; i++) {
    printf("%d ", c[i]);
    if ((i + 1) % 3 == 0)
      printf("\n");
  }

  exit(0);
  */
