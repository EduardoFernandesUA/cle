
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <time.h>

__global__ void dev_mergesort(uint32_t *a, uint32_t *b, uint32_t arrayLen,
                              uint32_t step) {
  uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  index *= step;

  // printf("BAH %d %d %d %d\n", index, blockIdx.x, blockDim.x, + threadIdx.x);

  if (index >= arrayLen) {
    return;
  }

  uint32_t halfStep = step >> 1;
  uint32_t i, k, l, r;

  k = index;
  l = index;
  r = index + halfStep;

  while (l < index + halfStep && r < index + step) {
    if (a[l] < a[r]) {
      b[k] = a[l];
      l++;
    } else {
      b[k] = a[r];
      r++;
    }
    k++;
  }
  while (l < index + halfStep) {
    b[k] = a[l];
    l++;
    k++;
  }
  while (r < index + step) {
    b[k] = a[r];
    r++;
    k++;
  }
  for (i = index; i < index + step; i++) {
    a[i] = b[i];
  }
}

__global__ void dev_bitonicsort1(uint32_t *n, uint32_t size, uint32_t step,
                                 uint32_t j) {
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size / 2) {
    return;
  }

  uint32_t j1 = j << 1;
  uint32_t imj = i % j;
  uint32_t idj = (i / j) * j1;

  uint32_t n1 = idj * j1 + imj;
  uint32_t n2 = idj * j1 + j1 - imj - 1;

  if (n[n1] > n[n2]) {
    uint32_t t = n[n1];
    n[n1] = n[n2];
    n[n2] = t;
  }
}

__global__ void dev_bitonicsort2(uint32_t *n, uint32_t size, uint32_t step,
                                 uint32_t j) {
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size / 2)
    return;

  uint32_t j1 = j << 1;

  uint32_t n1 = (i / j) * j1 + i % j;
  uint32_t n2 = n1 + j;

  if (n[n1] > n[n2]) {
    uint32_t t = n[n1];
    n[n1] = n[n2];
    n[n2] = t;
  }
}

void copynumbers(uint32_t *a, uint32_t *b, uint32_t len) {
  for (uint32_t i = 0; i < len; i++) {
    a[i] = b[i];
  }
}

void caps(uint32_t *a, uint32_t *b) {
  uint32_t temp;
  if (*a > *b) {
    temp = *a;
    *a = *b;
    *b = temp;
  }
}

// dir is 1 for ascending and -1 for descending
void caps(uint32_t *a, uint32_t *b, int dir) {
  uint32_t temp;
  if ((dir < 0 && *a > *b) || (dir > 0 && *a < *b)) {
    temp = *a;
    *a = *b;
    *b = temp;
  }
}

void host_bitonicsort2(uint32_t *numbers, uint32_t arrayLen) {
  uint32_t N = arrayLen;

  printf("\nHOST CODE\n");
  // first pass, groups of 2
  for (int i = 0; i < arrayLen; i += 2) {
    caps(&numbers[i], &numbers[i + 1], ((i % 4) - 1));
  }
  printf("\n\n");
  // second pass, groups of 2
  for (int i = 0; i < arrayLen; i += arrayLen / 2) {
    for (int j = i; j < i + arrayLen / 4; j++) {
      caps(&numbers[j], &numbers[j + arrayLen / 4], ((i % 8) - 2));
    }
  }
  printf("\n\n");

  // third pass
  for (int i = 0; i < arrayLen; i += 2) {
    printf("%d ", (i / 4) * 2 - 1);
    caps(&numbers[i], &numbers[i + 1], (i / 4) * 2 - 1);
  }
  printf("\n\n");
}

void host_bitonicsort(uint32_t *n, uint32_t size) {
  uint i, j, k;
  uint t_count = size / 2;
  uint log2size = -1;
  uint32_t a, b;

  // calculate thread count
  i = size;
  while (i != 0) {
    i >>= 1;
    log2size++;
  }

  uint32_t j0 = 1, j1 = 2, step = 0, jj0, jj1;
  for (j = 1; j <= log2size; j++) {
    for (i = 0; i < t_count; i++) {
      a = (i / j0) * j1 + i % j0;
      b = (i / j0) * j1 + j1 - i % j0 - 1;
      // printf("( %d, %d)\n", a, b);
      caps(&n[a], &n[b]);
    }

    // for (k = step; k > 0; k--) {
    for (k = 1; k <= step; k++) {
      for (i = 0; i < t_count; i++) {
        jj0 = j0 >> k;
        jj1 = j1 >> k;
        a = (i / jj0) * jj1 + i % jj0;
        b = a + jj0;
        caps(&n[a], &n[b]);
        // printf("_( %d, %d)\n", bah, bah + (j0 >> k));
      }
    }

    j0 <<= 1;
    j1 <<= 1;
    step += 1;
  }
}

void host_mergeSort(uint32_t *numbers, uint32_t arrayLen) {
  uint32_t i, k;
  uint32_t *ntemp = (uint32_t *)malloc(arrayLen * 4);

  // second pass
  uint32_t l, r, step, halfStep;
  for (step = 2; step <= arrayLen; step <<= 1) {
    halfStep = step >> 1;

    for (i = 0; i < arrayLen; i += step) {
      k = i;
      l = i;
      r = i + halfStep;
      while (l < i + halfStep && r < i + step) {
        if (numbers[l] < numbers[r]) {
          ntemp[k] = numbers[l];
          l++;
        } else {
          ntemp[k] = numbers[r];
          r++;
        }
        k++;
      }
      while (l < i + halfStep) {
        ntemp[k] = numbers[l];
        l++;
        k++;
      }
      while (r < i + step) {
        ntemp[k] = numbers[r];
        r++;
        k++;
      }
    }
    for (i = 0; i < arrayLen; i++) {
      numbers[i] = ntemp[i];
    }
  }

  free(ntemp);
}

void host_wikibitonicsort(uint32_t *n, uint32_t size) {
  int k, j, i, l, t;

  for (k = 2; k <= size; k *= 2) {
    for (j = k / 2; j > 0; j /= 2) {
      for (i = 0; i < size; i++) {
        l = i ^ j;
        if (l > i) {
          if (((i & j) == 0 && n[i] > n[l]) || ((i & j) != 0 && n[i] < n[l])) {
            t = n[i];
            n[i] = n[l];
            n[l] = t;
          }
        }
      }
    }
  }
}

int main(int argc, char *argv[]) {

  if (argc != 2) {
    printf("Usage: main_default.c <input_file.bin>");
    return 1;
  }

  uint i, err;
  clock_t clk;
  FILE *fd = fopen(argv[1], "rb");

  err = fseek(fd, 0, SEEK_END);
  uint64_t fileLen = ftell(fd) - 4;
  printf("File Size: %ld Bytes\n", fileLen);

  uint64_t arrayLen = fileLen / 4;
  // uint32_t arrayLen = 8;
  //  printf("Amount of numbers: %d\n", arrayLen);

  // Error bellow, the array is not generic for the size of the numbers, only 4
  // bytes now
  uint32_t *numbers = (uint32_t *)malloc(arrayLen * sizeof(uint32_t));
  uint32_t *numbers_dcopy = (uint32_t *)malloc(arrayLen * sizeof(uint32_t));

  // read numbers from the file
  i = 0;
  fseek(fd, 0, 0);
  do {
    err = fread(&numbers[i], sizeof(uint32_t), 1, fd);
    i++;
  } while (err);

  uint32_t *d_n, bytes = arrayLen * sizeof(uint32_t);
  hipMalloc(&d_n, bytes);
  hipMemcpy(d_n, numbers, bytes, hipMemcpyHostToDevice);

  int blockSize = 512;
  int numBlocks = (arrayLen / 2 + blockSize - 1) / blockSize;

  clk = clock();
  uint32_t j = 1, k;
  uint32_t nsteps = -1;
  i = arrayLen;
  while (i != 0) {
    i >>= 1;
    nsteps++;
  }
  printf("Log Amount = %d\n", nsteps);
  for (uint32_t step = 0; step < nsteps; step++) {
    dev_bitonicsort1<<<numBlocks, blockSize>>>(d_n, arrayLen, step, j);
    hipDeviceSynchronize();

    // printf("step2\n");
    for (k = 1; k <= step; k++) {
      dev_bitonicsort2<<<numBlocks, blockSize>>>(d_n, arrayLen, step, j >> k);
      hipDeviceSynchronize();
      // printf("break\n");
    }

    j <<= 1;
    // printf("WWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWWW\n");
  }
  clk = clock() - clk;
  printf("Device bitonic Sort: %fs\n", ((double)clk) / CLOCKS_PER_SEC);
  hipMemcpy(numbers_dcopy, d_n, bytes, hipMemcpyDeviceToHost);

  uint32_t *hostnumbers = (uint32_t *)malloc(arrayLen * sizeof(uint32_t));
  copynumbers(hostnumbers, numbers, arrayLen);
  clk = clock();
  host_mergeSort(hostnumbers, arrayLen);
  clk = clock() - clk;
  printf("Host Merge Sort: %fs\n", ((double)clk) / CLOCKS_PER_SEC);

  copynumbers(hostnumbers, numbers, arrayLen);
  clk = clock();
  host_bitonicsort(hostnumbers, arrayLen);
  clk = clock() - clk;
  printf("Host Wiki Bitonic Sort: %fs\n", ((double)clk) / CLOCKS_PER_SEC);
  for (i = 0; i < arrayLen; i++) {
    // printf("%d\n", hostnumbers[i]);
  }

  // Verify device solution agains host
  // for (i = 0; i < arrayLen; i++) {
  //   printf("%8d %8d", hostnumbers[i], numbers_dcopy[i]);
  //   if (hostnumbers[i] != numbers_dcopy[i]) {
  //     printf("  WRONG SOLUTION\n");
  //   } else {
  //     printf("\n");
  //   }
  // }

  // printf("\nSorted\n");
  // for (i = 0; i < arrayLen; i++) {
  //   printf("%u\n", numbers[i]);
  // }

  hipFree(d_n);
  free(hostnumbers);
  free(numbers);

  return 0;
}
